#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include "ContourCUDA.h"

__global__ void countEdges(float *vertexes, int nRows, int nCols, int *numExpectedPoints, float level)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x; //found using the dimensions, based on the file given on the in-class materials page
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = y * gridDim.x * blockDim.x + x;

  int bound = ((nRows * nCols) - 1) - nCols; 
  if (index <= bound && ((index + 1) % nCols != 0)) //check if row below top row, and column before last column
  {
    //local values to determine how many edges to expect from the grid
    int count = 0;
    int nAbove = 0;
    int nBelow = 0;

    //each point to check
    float bottomL = vertexes[index];
    float bottomR = vertexes[index + 1];
    float topL = vertexes[index + nCols];
    float topR = vertexes[index + nCols + 1];

    //check if values are above or below the level, add accordingly
    if (bottomL > level)
    {
      nAbove++;
    }
    else
    {
      nBelow++;
    }
    if (bottomR > level)
    {
      nAbove++;
    }
    else
    {
      nBelow++;
    }
    if (topL > level)
    {
      nAbove++;
    }
    else
    {
      nBelow++;
    }
    if (topR > level)
    {
      nAbove++;
    }
    else
    {
      nBelow++;
    }

    //calculate number of expected edges based on how many vertices were below or above the desired level
    if (nAbove == 3 && nBelow == 1)
    {
      count = 1;
    }
    else if (nAbove == 1 && nBelow == 3)
    {
      count = 1;
    }
    else if (nAbove == 2 && nBelow == 2)
    {
      count = 2;
    }
    else
    {
      count = 0;
    }

    atomicAdd(numExpectedPoints, count); //add to the number of expected edges total
  }
}

__global__ void computeKernel(float *vertexes, int nRows, int nCols, int level, int *edgeCount, vec2 *actualEdgePoints, int *buf_location)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x; 
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = y * gridDim.x * blockDim.x + x; //the index in the vertex array, acquired by multiplying the dimensions of each block and the threads within those blocks

  int bound = ((nRows * nCols) - 1) - nCols; //we do not want to check the top row of the grid or the index at the farthest right
  if (index <= bound && ((index % nCols) != nCols - 1)) //check if row below top row, and column before last column
  {
    //each point to check
    float bottomL = vertexes[index];
    float bottomR = vertexes[index + 1];
    float topL = vertexes[index + nCols];
    float topR = vertexes[index + nCols + 1];

    int loc; //the location of our index in the actualEdgePoints array so that we do not overlap edge points
    bool vertfound = false; //if we have found one vertex already
    int count = 0; //the number of vertexes we have found so far pertaining to one edge
    float x_coord = -1.0;
    float y_coord = -1.0;

    //check for missing data and return if found missing
    if (bottomL == -9999 || bottomR == -9999 || topL == -9999 || topR == -9999)
    {
      return; //do not check
    }

    //check every corner of the square starting from the bottom line, to right vertical, top horizontal, then left vertical
    if ((bottomL <= level && level <= bottomR) || (bottomL > level && level > bottomR)) //if the level is between the two points, not dependent on which corner is greater
    {
      if (bottomL <= level && level <= bottomR) //if the bottom right is greater
      {
        float f = (level - bottomL) / (bottomR - bottomL); //using the function given to find the coordinate between points
        x_coord = (1.0 - f) * (index % nCols) + f * ((index + 1) % nCols); //use that percentage and attribute it to x and y values, depending on which part of the square we are checking
        y_coord = (float)(index / nCols); //use the normal y coordinate
      }
      else if (bottomL > level && level > bottomR) //bottom left is greater, so the function is switched backwards
      {
        float f = (level - bottomR) / (bottomL - bottomR);
        x_coord = (1.0 - f) * ((index + 1) % nCols) + f * (index % nCols);
        y_coord = (float)(index / nCols);
      }
      if (!vertfound) //we have not found a vertice already, this is the first point of our edge
      {
        loc = atomicAdd(buf_location, 2); //get the index to add this vertex coordinate set to the actualEdgePoint array
        vertfound = true; //set to true so that we know we are on our second vertex of a certain edge
      }
      actualEdgePoints[loc + count][0] = x_coord; //set the coordinates of the vertex
      actualEdgePoints[loc + count][1] = y_coord;
      count++; //add to know how many vertices we have added so far
      if (count == 2) //checks if we have completed our edge with 2 vertices, reset the edge count
      {
        vertfound = 0;
        count = 0;
        atomicAdd(edgeCount, 1); //add to the total number of edges that we have
      }
    }

    //repeat
    if ((bottomL <= level && level <= topL) || (bottomL > level && level > topL))
    {
      if (bottomL <= level && level <= topL)
      {
        float f = (level - bottomL) / (topL - bottomL);
        x_coord = (float)(index % nCols);
        y_coord = (1.0 - f) * (index / nCols) + f * ((index + nCols) / nCols);
      }
      else if (bottomL > level && level > topL)
      {
        float f = (level - topL) / (bottomL - topL);
        x_coord = (float)(index % nCols);
        y_coord = (1.0 - f) * ((index + nCols) / nCols) + f * (index / nCols);
      }
      if (!vertfound)
      {
        loc = atomicAdd(buf_location, 2);
        vertfound = true;
      }
      actualEdgePoints[loc + count][0] = x_coord;
      actualEdgePoints[loc + count][1] = y_coord;
      count++;
      if (count == 2)
      {
        vertfound = 0;
        count = 0;
        atomicAdd(edgeCount, 1);
      }
    }

    if ((topR <= level && level <= topL) || (topR > level && level > topL))
    {
      if (topR <= level && level <= topL)
      {
        float f = (level - topR) / (topL - topR);
      x_coord = (1.0 - f) * ((index + nCols + 1) % nCols) + f * ((index + nCols) % nCols);
      y_coord = (float)((index + nCols) / nCols);
      }
      else if (topR > level && level > topL)
      {
        float f = (level - topL) / (topR - topL);
      x_coord = (1.0 - f) * ((index + nCols) % nCols) + f * ((index + nCols + 1) % nCols);
      y_coord = (float)((index + nCols) / nCols);
      }
      if (!vertfound)
      {
        loc = atomicAdd(buf_location, 2);
        vertfound = true;
      }
      actualEdgePoints[loc + count][0] = x_coord;
      actualEdgePoints[loc + count][1] = y_coord;
      count++;
      if (count == 2)
      {
        vertfound = 0;
        count = 0;
        atomicAdd(edgeCount, 1);
      }
    }

    if ((topR <= level && level <= bottomR) || (topR > level && level > bottomR))
    {
      if (topR <= level && level <= bottomR)
      {
        float f = (level - topR) / (bottomR - topR);
      x_coord = (float)((index + 1) % nCols);
      y_coord = (1.0 - f) * ((index + nCols + 1) / nCols) + f * ((index + 1) / nCols);
      }
      else if (topR > level && level > bottomR)
      {
         float f = (level - bottomR) / (topR - bottomR);
      x_coord = (float)((index + 1) % nCols);
      y_coord = (1.0 - f) * ((index + 1) / nCols) + f * ((index + nCols + 1) / nCols);
      }

      if (!vertfound)
      {
        loc = atomicAdd(buf_location, 2);
        vertfound = true;
      }
      actualEdgePoints[loc + count][0] = x_coord;
      actualEdgePoints[loc + count][1] = y_coord;
      count++;
      if (count == 2)
      {
        vertfound = 0;
        count = 0;
        atomicAdd(edgeCount, 1);
      }
    }
  }
}

int expectedEdgesKernel(float *vertexes, int nRows, int nCols, float level)
{
  float *dev_varray;                               //device vertex array buffer to copy
  int vert_size = (nRows * nCols) * sizeof(float); //size of vertex array to copy to gpu

  int *dev_count;          //expected edge device count variable to copy to gpu
  int zero = 0;            //start the device count at 0
  int *host_count = &zero; //host count to copy gpu value back to cpu

  hipMalloc((void**)&dev_varray, vert_size);  //allocate size to hold the vertex array in gpu
  hipMalloc((void**)&dev_count, sizeof(int)); //allocate one int variable on gpu to hold edge count

  hipMemcpy(dev_varray, vertexes, vert_size, hipMemcpyHostToDevice);    //copy vertexValues to the gpu in dev_varray
  hipMemcpy(dev_count, host_count, sizeof(int), hipMemcpyHostToDevice); //copy edge count to gpu starting at 0

  dim3 block(16, 16);                                                          //placeholder size for blocks only optimized for warps
  dim3 grid((nRows + block.x - 1) / block.x, (nCols + block.y - 1) / block.y); //launch grid based on size of vertexValues divided by block thread size

  countEdges<<<grid, block>>>(dev_varray, nRows, nCols, dev_count, level); //call kernel to count expected edges
  hipDeviceSynchronize();                                                 //barrier

  hipMemcpy(host_count, dev_count, sizeof(int), hipMemcpyDeviceToHost); //copy device count back to host count to pass back

  hipFree(dev_varray); //free gpu vertex array
  hipFree(dev_count);  //free device count

  return *host_count;
}

int actualEdgesKernel(float *vertexes, int nRows, int nCols, float level, int numExpectedPoints, vec2 *buffer)
{
  float *dev_varray;                               //device vertex array buffer to copy
  int vert_size = (nRows * nCols) * sizeof(float); //size of vertex array to copy to gpu

  int *dev_count;          //actual edges device count variable to copy to gpu
  int zero = 0;            //start the device count at 0
  int *host_count = &zero; //host count to copy gpu value back to cpu
  int *buf_location;       //index of the buffer that the coordinates should be placed at so edges are correct

  hipMalloc(&dev_varray, vert_size);     //allocate size to hold the vertex array in gpu
  hipMalloc(&dev_count, sizeof(int));    //allocate one int variable on gpu to hold actual edge count
  hipMalloc(&buf_location, sizeof(int)); //allocate index of buffer we are writing coordinates to

  vec2 *dev_buffer;                                               //allocate buffer to hold points of actual edges calculated
  hipMalloc(&dev_buffer, 2 * numExpectedPoints * sizeof(float)); //two points for each edge calculated

  hipMemcpy(dev_varray, vertexes, vert_size, hipMemcpyHostToDevice);       //copy vertexValues to the gpu in dev_varray
  hipMemcpy(dev_count, host_count, sizeof(int), hipMemcpyHostToDevice);    //copy edge count to gpu starting at 0
  hipMemcpy(buf_location, host_count, sizeof(int), hipMemcpyHostToDevice); //copy buffer index location to gpu
  hipMemcpy(dev_buffer, buffer, 2 * numExpectedPoints * sizeof(float), hipMemcpyHostToDevice);

  dim3 block(16, 16);                                                          //placeholder size for blocks only optimized for warps
  dim3 grid((nRows + block.x - 1) / block.x, (nCols + block.y - 1) / block.y); //launch grid based on size of vertexValues divided by block thread size

  computeKernel<<<grid, block>>>(dev_varray, nRows, nCols, level, dev_count, dev_buffer, buf_location); //compute actual number of edges in vertex array
  hipDeviceSynchronize();                                                                              //barrier

  hipMemcpy(host_count, dev_count, sizeof(int), hipMemcpyDeviceToHost);                        //copy back actual number of edges calculated
  hipMemcpy(buffer, dev_buffer, 2 * numExpectedPoints * sizeof(float), hipMemcpyDeviceToHost); //copy the actual edges from the gpu to the actual_edge_buffer on the cpu to then fill lines

  hipFree(dev_varray); //free gpu vertex array
  hipFree(dev_count);  //free device count
  hipFree(dev_buffer); //free gpu actual edge buffer

  return *host_count;
}
